#include "hip/hip_runtime.h"
﻿/*
* This is a class that holds the methods used for the epiblaster GPU Kernel
* Author: Lance Hartman
* Date: 7/20/2023
*/

#include "hip/hip_runtime.h"
#include ""

#include "kernel.cuh"

#include <algorithm>

__device__ void Matrix::print_matrix() {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f, ", *(elements + i * width + j));
        }
        printf("\n");
    }
    printf("\n");
}

__device__ double Matrix::mean() {
    double total = 0;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            total += *(elements + i * width + j);
        }
    }
    return total / (width * height);
}

__device__ double Matrix::standard_dev() {
    double total = 0;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            total += __CUDA_RUNTIME_API_H__::pow((*(elements + i * width + j) - mean()), 2);
        }
    }
    return sqrt((double)total / ((width * height) - 1));
}

/**
* Matrix helper method cuz I couldn't figure out how to properly overload the operator. . .
* @param first First matrix
* @param other Matrix being subtracted onto the first matrix
* @returns First matrix - Second matrix
*/
__device__ Matrix subtract_matrices(const Matrix& first, const Matrix& other) {
    if (first.width != other.width || first.height != other.height) {
        printf("Error: cannot subtract matrixes with different sizes");
        return Matrix{};
    }
    else {
        Matrix ret = { first.width, first.height, first.elements };
        for (int i = 0; i < first.height; i++) {
            for (int j = 0; j < first.width; j++) {
                ret.elements[i * ret.width + j] -= other.elements[i * other.width + j];
            }
        }
        return ret;
    }
}

/**
* Method to transpose a matrix
* @param A Matrix to get transposed
* @returns A transposed copy of the matrix
*/
__device__ Matrix transpose(Matrix A)
{
    Matrix ret;
    ret.width = A.height;
    ret.height = A.width;
    ret.elements = new double[sizeof(A.elements) / sizeof(A.elements[0])];
    for (int i = 0; i < A.height; i++) {
        for (int j = 0; j < A.width; j++) {
            // M(row, col) = *(M.elements + row * M.width + col)
            ret.elements[j * ret.width + i] = A.elements[i * A.width + j];
        }
    }
    return ret;
}

/**
* Cross product method (sort of)
* This replicates the R crossprod function which actually returns one matrix transposed multiplied by another matrix
* @param A One of the matrixes we will be using in our cross product
* @param B The other matrix we will be using
* @returns A matrix representation of the cross product (in this case: transposed matrix multiplied by other matrix)
*/
__device__ Matrix cross_product(Matrix A, Matrix B)
{
    Matrix C;
    Matrix a_transposed = transpose(A); //Transposed copy of A

    //Now we need to multiply the two matrices together
    if (a_transposed.width != B.height) {
        printf("Matrix sizes are not compatable!");
        return;
    }

    C.width = a_transposed.height;
    C.height = B.width;
    C.elements = new double[a_transposed.height * B.width];

    //First for loop -> per a_transposed row
    for (int k = 0; k < a_transposed.height; k++) {
        //Second for loop -> per B column
        for (int i = 0; i < B.width; i++) {
            //This is where we should determine which variable we are adding into
            double ret_element = 0;
            //Third for loop -> per B row
            for (int j = 0; j < B.height; j++) {
                // j should be the same for both a_transposed and B (if sizes are compatable)
                // M(row, col) = *(M.elements + row * M.width + col)
                ret_element += a_transposed.elements[k * a_transposed.width + j] * B.elements[j * B.width + i];
            }
            C.elements[k * C.width + i] = ret_element;
        }
    }
    return C;
}

/**
* Method to calculate the chunk range
* @param idx
* @param n
* @param chunk
* @returns Range struct holding chunk range info
*/
__device__ Range ithChunk(int idx, int n, int chunk)
{
    int start = (idx - 1) * chunk;
    if (idx < 0 || start > n) return Range{0,0}; // Should not happen!
    else return Range{start, (int)__CUDA_RUNTIME_H__::fmin((double)(idx * chunk), (double)n)}; //Make sure no rounding
}

/**
* Method to scale a matrix according to the R "scale()" function
* KEEP IN MIND SCALING IN CPP IS DIFFERENT THAN R
* @param A Matrix to be scaled
* @returns The scaled matrix
*/
__device__ Matrix scale(Matrix A){
    Matrix scaled = {
        A.width,
        A.height,
        new double[sizeof(A.elements)]
    };
    for (int i = 0; i < scaled.height; i++) {
        for (int j = 0; j < scaled.width; j++) {
            scaled.elements[i * scaled.width + j] = (*(A.elements + i * A.width + j) - A.mean()) / A.standard_dev();
        }
    }
    return scaled;
}

/**
* Calculates the Pearson Correlation Coefficient of two matrices
* @param A The first of the two matrixes
* @param B The second of the two matrixes
* @returns The correlation matrix
*/
__device__ Matrix getcor(Matrix A, Matrix B)
{
    if (A.height != B.height) {
        printf("A and B have different row numbers!");
        return Matrix{};
    }
    else {
        Matrix Abar = scale(A);
        //TODO issue with code below:
        for (int i = 0; i < Abar.height; i++) {
            for (int j = 0; j < Abar.width; j++) {
                Abar.elements[i * Abar.width + j] = Abar.elements[i * Abar.width + j]
                                                    * -1 * __CUDA_RUNTIME_H__::sqrt((double)(1.0 / (Abar.height - 1)));
            }
        }

        Matrix Bbar = scale(B);
        for (int i = 0; i < Bbar.height; i++) {
            for (int j = 0; j < Bbar.width; j++) {
                Bbar.elements[i * Bbar.width + j] = *(Bbar.elements + i * Abar.width + j)
                                                    * -1 * __CUDA_RUNTIME_H__::sqrt((double)(1.0 / (Bbar.height - 1)));
            }
        }
        return cross_product(Abar, Bbar);
    }
}

/**
Matrix A -> Transpose
Matrix B -> Normal
Multiply them together

      ztest <- (getcor(A = as.matrix(case[, ithChunk(as.numeric(i), nSNP, chunk), drop = FALSE]),
                       B = as.matrix(case[, ithChunk(as.numeric(j), nSNP, chunk), drop = FALSE]),
                       method = "pearson")
                -
                  getcor(A = as.matrix(control[, ithChunk(as.numeric(i), nSNP, chunk), drop = FALSE]),
                         B = as.matrix(control[, ithChunk(as.numeric(j), nSNP, chunk), drop = FALSE]),
                         method = "pearson") )  /  sd_tot
      index <- which(abs(ztest) >= zthres, arr.ind = TRUE)

      ifelse(i==j,
             WriteSnpPairs_sym,
             WriteSnpPairs)(Zmatrix = ztest, indexArr = index,
                            outfile = OUT)

*/
__global__ void EpiScanKernel(Matrix genotype_data, 
                              Matrix phenotype_data, 
                              double* zpthres, 
                              int* chunksize
                             ) {
    printf("-----------KERNEL ACTIVATED-------------\n");
    
    //Check to make sure same number of cases for genotype and phenotype
    if (genotype_data.height != phenotype_data.height) {
        printf("A and B do not have the same number of elements. Please check your data!");
        return;
    }

    //Check to make sure that the chunksize isn't greater than the width of the matrix
    if (genotype_data.width > *chunksize) *chunksize = genotype_data.width;
    

    /*
    A.print_matrix();
    printf("\n");
    B.print_matrix();
    printf("\n");
    */
    /*
    Matrix C = cross_product(A, B);

    C.print_matrix();

    Matrix scaled = scale(A);
    scaled.print_matrix();

    Matrix corr = getcor(A, B);
    corr.print_matrix();
    */
    printf("-----------KERNEL FINISHED-------------\n");
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE************
hipError_t EpiScan(const Matrix A, 
                    const Matrix B, 
                    const double zpthres, 
                    const int chunksize
                   ) {
    Matrix d_A = {};
    Matrix d_B = {};
    double* d_zpthres;
    int* d_chunksize;
    hipError_t cudaStatus;

    printf("EpiScan called!\n");

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Load A to device memory
    size_t size = A.width * A.height * sizeof(double); //Calculate the total amount of memory to allocate for matrix A
    d_A.width = A.width;
    d_A.height = A.height;
    cudaStatus = hipMalloc(&d_A.elements, size); //Allocate the data on the CUDA device
    if (cudaStatus != hipSuccess) {
        printf("d_A hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_A.elements, A.elements, size,
        hipMemcpyHostToDevice); //Copy the memory stored in the Matrix struct into the allocated memory
    if (cudaStatus != hipSuccess) {
        printf("d_A hipMemcpy failed!");
        goto Error;
    }

    // Load B to device memory
    size = B.width * B.height * sizeof(double); //Calculate the total amount of memory to allocate for matrix A
    d_B.width = B.width;
    d_B.height = B.height;
    cudaStatus = hipMalloc(&d_B.elements, size); //Allocate the data on the CUDA device
    if (cudaStatus != hipSuccess) {
        printf("d_B hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_B.elements, B.elements, size,
        hipMemcpyHostToDevice); //Copy the memory stored in the Matrix struct into the allocated memory
    if (cudaStatus != hipSuccess) {
        printf("d_B hipMemcpy failed!");
        goto Error;
    }

    //Load zpthres to memory device
    cudaStatus = hipMalloc((void**) & d_zpthres, sizeof(double));
    if (cudaStatus != hipSuccess) {
        printf("d_zpthres hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_zpthres, &zpthres, sizeof(double),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("d_zpthres hipMemcpy failed!");
        goto Error;
    }

    //Load chunksize to memory device
    cudaStatus = hipMalloc((void**)&d_chunksize, sizeof(int));
    if (cudaStatus != hipSuccess) {
        printf("d_chunksize hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(d_chunksize, &chunksize, sizeof(int),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("d_chunksize hipMemcpy failed!");
        goto Error;
    }

    printf("Memory Allocated!\n");

    // Invoke kernel

    //struct hipDeviceProp_t properties;
    //hipGetDeviceProperties(&properties, device);
    //cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
    //cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

    dim3 dimBlock(16, 16);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    EpiScanKernel <<<dimGrid, dimBlock>>> (d_A, d_B, d_zpthres, d_chunksize);

    hipDeviceSynchronize();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "EpiScanKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(d_zpthres);
    hipFree(d_chunksize);

Error:
    hipFree(&d_A);
    hipFree(&d_B);
    hipFree(d_zpthres);
    hipFree(d_chunksize);

    return cudaStatus;
}
